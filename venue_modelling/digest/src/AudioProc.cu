#include "hip/hip_runtime.h"
#include "AudioProc.h"

//keep these in here as compiler can get confused and not know where to look for these if using g++ for header file
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void combine(short int *d_monoBuff, short int *d_left_in, short int *d_right_in, float abs, int delay_l, int delay_r, int n){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    //thread id must be less than array length
    if (tid < n){
        short int sample_l = 0;
        //if larger than delay start reading the audio
        if (tid >= delay_l){
            sample_l = d_left_in[tid - delay_l];
            sample_l *= abs; // absorption loss
            sample_l = sample_l / 2; //average of both audio streams for combining to mono
        }
        d_monoBuff[tid] += sample_l; 

        short int sample_r = 0;
        //if larger than delay start reading the audio
        if (tid >= delay_r){
            sample_r = d_right_in[tid - delay_r];
            sample_r = sample_r * abs; // absorption loss
            sample_r = sample_r / 2; //average of both audio streams for combining to mono
        }
        d_monoBuff[tid] += sample_r; 
    }
}

AudioProc::AudioProc(/* args */){

}

AudioProc::~AudioProc(){
    
    //will need to use the destructors for audio pointers
    //or use shared smart pointers for the audio
    //wait destruct in main loop right?
    //or it will be a double free

    //also make room arr smart pointer

    deviceFreeMono();
    deviceFreeStereo();
    hostFreeMono();
    hostFreeStereo();
}

void AudioProc::process(int x, int y, int z){

    //means that everything is set (audioleng, smaple rate, buffs) ie fileread check bool
    if (!fileRead){

        std::cout << "process was called before a file was read in!";
        return;
    }

    //reset monobuff then copy over to device
    for (int i = 0; i < m_audioLeng; i++){
        monoBuff[i] = 0;
    }
    hipMemcpy(d_monoBuff, monoBuff, sizeof(short int) * m_audioLeng, hipMemcpyHostToDevice); //copies 

    //populate room buffer arrays
    for (int r = 0; r < m_rooms; r++){

        //get virtual coords of point
        int room_x = x + (m_roomArr[r].pos_x * m_xLength);
        int room_y = y + (m_roomArr[r].pos_y * m_yLength);
        int room_z = z + (m_roomArr[r].pos_z * m_zLength);
        
        //if room mirrored in dimension, add extra difference to get to the actual coord
        if (m_roomArr[r].mirrored_x){
            room_x = room_x + (2 * ((m_xLength/2) - x));
        }

        if (m_roomArr[r].mirrored_y){
            room_y = room_y + (2 * ((m_yLength/2) - y));
        }

        if (m_roomArr[r].mirrored_z){
            room_z = room_z + (2 * ((m_zLength/2) - z));
        }

        //dist to left
        float dist_L = get_dist(room_x, room_y, room_z, m_speaker_L);
        //dist to right
        float dist_R = get_dist(room_x, room_y, room_z, m_speaker_R);

        //time delays in samples
        int delay_L_samp = (dist_L/343) * m_sampleRate;
        int delay_R_samp = (dist_R/343) * m_sampleRate;

        // Executing kernel 
        int block_size = 256;
        int grid_size = ((m_audioLeng + block_size) / block_size); //add extra 256 to N so that when dividing it will round down to > required threads
        combine<<<grid_size,block_size>>>(d_monoBuff, d_left_in, d_right_in, m_roomArr[r].totalAbs / m_gain, delay_L_samp, delay_R_samp, m_audioLeng);
    }

    //synchronize?? (wait for gpu calls to stop)

    hipMemcpy(monoBuff, d_monoBuff, sizeof(short int) * m_audioLeng, hipMemcpyDeviceToHost);

}

void AudioProc::init(std::string username){

    //move all this somewhere else and just init all this in the constructor then call bake

    //dimensions
    m_xLength = selectInt("SELECT xLength FROM userTable WHERE username = '" + username + "'", "xLength");
    m_yLength = selectInt("SELECT yLength FROM userTable WHERE username = '" + username + "'", "yLength");
    m_zLength = selectInt("SELECT zLength FROM userTable WHERE username = '" + username + "'", "zLength");

    //resolution and order 
    m_resolution = selectInt("SELECT resolution FROM userTable WHERE username = '" + username + "'", "resolution");
    m_order = selectInt("SELECT reflections FROM userTable WHERE username = '" + username + "'", "reflections");

    //absorption coeficients
    m_coefs_x.neg = selectFloat("SELECT xNeg FROM userTable WHERE username = '" + username + "'", "xNeg");
    m_coefs_x.pos = selectFloat("SELECT xPos FROM userTable WHERE username = '" + username + "'", "xPos");
    m_coefs_y.neg = selectFloat("SELECT yNeg FROM userTable WHERE username = '" + username + "'", "yNeg");
    m_coefs_y.pos = selectFloat("SELECT yPos FROM userTable WHERE username = '" + username + "'", "yPos");
    m_coefs_z.neg = selectFloat("SELECT zNeg FROM userTable WHERE username = '" + username + "'", "zNeg");
    m_coefs_z.pos = selectFloat("SELECT zPos FROM userTable WHERE username = '" + username + "'", "zPos");

    bake();
}

//calculate distance between two points
float AudioProc::get_dist(int x, int y, int z, point speaker){

    float dist = sqrt(pow(x-speaker.x, 2)+pow(y-speaker.y, 2)+pow(z-speaker.z, 2)) / m_resolution;

    return dist;
}

void AudioProc::bake(){

    //init dimensions and point num
    m_xLength = m_xLength * m_resolution;
    m_yLength = m_yLength * m_resolution;
    m_zLength = m_zLength * m_resolution;
    m_points = m_xLength * m_yLength * m_zLength;

    //init rooms
    //reflections is extra on the border, this adds both sides of border plus middle to get total dimension length
    int rooms_x = (m_order * 2) + 1;
    int rooms_y = (m_order * 2) + 1;
    int rooms_z = (m_order * 2) + 1;

    //total rooms is x*y*z
    m_rooms = rooms_x * rooms_y * rooms_z;

    m_roomArr = new room[m_rooms]; 

    std::cout << "rooms: " << m_rooms << std::endl;

    m_gain = ceil(((rooms_x*m_coefs_x.neg*m_coefs_x.pos)+(rooms_y*m_coefs_y.neg*m_coefs_y.pos)+(rooms_y*m_coefs_y.neg*m_coefs_y.pos))/3);

    //calculate total abs coeffs and check if room is mirrored
    int r = 0;
    for(int y = 0; y < rooms_y; y++){
        for(int z = 0; z < rooms_z; z++){
            for(int x = 0; x < rooms_x; x++){

                m_roomArr[r].pos_x = x - m_order;
                m_roomArr[r].pos_y = y - m_order;
                m_roomArr[r].pos_z = z - m_order;

                int absolute_x = abs(m_roomArr[r].pos_x);
                int absolute_y = abs(m_roomArr[r].pos_y);
                int absolute_z = abs(m_roomArr[r].pos_z);

                if (m_roomArr[r].pos_x % 2 != 0){
                    //is mirrored in this axis
                    m_roomArr[r].mirrored_x = 1;

                    float extra_coef = 1;
                    //extra coef on the end changes whether going in positive or negative direction
                    if (m_roomArr[r].pos_x < 0){
                        extra_coef *= m_coefs_x.pos;
                    } else {
                        extra_coef *= m_coefs_x.neg;
                    }

                    m_roomArr[r].totalAbs *= pow(m_coefs_x.neg, (absolute_x - 1) / 2) * pow(m_coefs_x.pos, (absolute_x - 1) / 2) * extra_coef;
                } else {
                    m_roomArr[r].totalAbs *= pow(m_coefs_x.neg, absolute_x / 2) * pow(m_coefs_x.pos, absolute_x / 2);
                }

                if (m_roomArr[r].pos_y % 2 != 0){
                    m_roomArr[r].mirrored_y = 1;

                    float extra_coef = 1;
                    //extra coef on the end changes whether going in positive or negative direction
                    if (m_roomArr[r].pos_y < 0){
                        extra_coef *= m_coefs_y.pos;
                    } else {
                        extra_coef *= m_coefs_y.neg;
                    }

                    m_roomArr[r].totalAbs *= pow(m_coefs_y.neg, (absolute_y - 1) / 2) * pow(m_coefs_y.pos, (absolute_y - 1) / 2) * extra_coef;
                } else {
                    m_roomArr[r].totalAbs *= pow(m_coefs_y.neg, absolute_y / 2) * pow(m_coefs_y.pos, absolute_y / 2);
                }

                if (m_roomArr[r].pos_z % 2 != 0){
                    m_roomArr[r].mirrored_z = 1;

                    float extra_coef = 1;
                    //extra coef on the end changes whether going in positive or negative direction
                    if (m_roomArr[r].pos_z < 0){
                        extra_coef *= m_coefs_z.pos;
                    } else {
                        extra_coef *= m_coefs_z.neg;
                    }

                    m_roomArr[r].totalAbs *= pow(m_coefs_z.neg, (absolute_z - 1) / 2) * pow(m_coefs_z.pos, (absolute_z - 1) / 2) * extra_coef;
                } else {
                    m_roomArr[r].totalAbs *= pow(m_coefs_z.neg, absolute_z / 2) * pow(m_coefs_z.pos, absolute_z / 2);
                }

                r++;
            }
        }
    }

    //speaker positioning
    m_speaker_L.x = m_xLength/4;
    m_speaker_L.y = m_yLength/2;
    m_speaker_L.z = 1;

    m_speaker_R.x = (m_xLength/4) * 3;
    m_speaker_R.y = m_yLength/2;
    m_speaker_R.z = 1;
}


//gets and sets
int AudioProc::getAudioLeng(){

    return m_audioLeng;
}

void AudioProc::setAudioLeng(int val){

    m_audioLeng = val;
}

int AudioProc::getSampleRate(){
    return m_sampleRate;
}

void AudioProc::setSampleRate(int val){
    m_sampleRate = val;
}

int AudioProc::getFrameSize(){
    return m_frameSize;
}

void AudioProc::setFrameSize(int val){
    m_frameSize = val;
}

point AudioProc::getSpeakerPosition(char side){
    if (side == 0){
        return m_speaker_L;
    } else {
        return m_speaker_R;
    }
}

dimensions AudioProc::getDimensions(){

    dimensions dims;

    dims.xLength = m_xLength;
    dims.yLength = m_yLength;
    dims.zLength = m_zLength;

    return dims;
}


//host and device memory untilities

void AudioProc::copyStereoToDevice(){

    hipMemcpy(d_left_in, left_in, sizeof(short int) * m_audioLeng, hipMemcpyHostToDevice);
    hipMemcpy(d_right_in, right_in, sizeof(short int) * m_audioLeng, hipMemcpyHostToDevice);
}

//stereo buffers dev and host
void AudioProc::deviceMallocStereo(){

    //need checks audioleng is set

    hipMalloc((void**)&d_left_in, sizeof(short int) * m_audioLeng);
    hipMalloc((void**)&d_right_in, sizeof(short int) * m_audioLeng);
}

void AudioProc::deviceFreeStereo(){
    hipFree(d_left_in);
    hipFree(d_right_in);
}

void AudioProc::hostMallocStereo(){

    //need checks audioleng is set

    left_in = new short int [m_audioLeng];
    right_in = new short int [m_audioLeng];
}

void AudioProc::hostFreeStereo(){

    delete [] left_in;
    delete [] right_in;
}

//mono buffer dev and host
void AudioProc::deviceMallocMono(){

    //need checks audioleng is set

    hipMalloc((void**)&d_monoBuff, sizeof(short int) * m_audioLeng);
}

void AudioProc::deviceFreeMono(){
    hipFree(d_monoBuff);
}

void AudioProc::hostMallocMono(){

    //need checks audioleng is set
    
    monoBuff = new short int [m_audioLeng];
}

void AudioProc::hostFreeMono(){
    delete [] monoBuff;
}
